
// HEADERS
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void arradd(int *x, int *y, int *z) // kernel definition
{
    int id = blockIdx.x;
    /* blockIdx.x gives the respective block id which starts from 0 */
    z[id] = x[id] + y[id];
}

//ENTRY POINT OF MAIN FUNCTION
int main(void)  
{
    int arr1[6];
    int arr2[6];
    int arr3[6];
    int *gpu_arr1, *gpu_arr2, *gpu_arr3;
    int i;


    printf("\n putting values in first array\n");
    for (i = 0; i < sizeof(arr1)/sizeof(arr1[0]) ; i++)
    {
        arr1[i] = rand() % 1024;
    }
    printf("\n putting values in second array\n");
    for (i = 0; i < sizeof(arr1)/sizeof(arr1[0]) ; i++)
    {
        arr2[i] = rand();
    }

    printf("\nprinting values of first array\n");
    for (i = 0; i < sizeof(arr2)/sizeof(arr2[0]); i++)
    {
        printf("arr1[%d] = %d\n", i, arr1[i]);
    }

    printf("\nprinting values of second array\n");
    for (i = 0; i < sizeof(arr2)/sizeof(arr2[0]); i++)
    {
        printf("arr2[%d] = %d\n", i, arr2[i]);
    }


    /* cudaMalloc() allocates memory from Global memory on GPU */
    hipMalloc((void **)&gpu_arr1, 6 * sizeof(int)); 
    hipMalloc((void **)&gpu_arr2, 6 * sizeof(int));
    hipMalloc((void **)&gpu_arr3, 6 * sizeof(int));

    /* cudaMemcpy() copies the contents from destination to source. Here destination is GPU(gpu_arr1, gpu_arr2)
    and source is CPU(a,b) */
    hipMemcpy(gpu_arr1, arr1, 6 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(gpu_arr2, arr2, 6 * sizeof(int), hipMemcpyHostToDevice);

    /* call to kernel. Here 6 is number of blocks, 1 is the number of threads per block and gpu_arr1, gpu_arr2, gpu_arr3 are
    the arguments */
    arradd<<<6, 1>>>(gpu_arr1, gpu_arr2, gpu_arr3);
    
    /* Here we are copying content from GPU(Device) to CPU(Host) */
    hipMemcpy(arr3, gpu_arr3, 6 * sizeof(int), hipMemcpyDeviceToHost);
    printf("\nSum of two arrays:\n ");
    for (i = 0; i < 6; i++)
    {
        printf("%d\t", arr3[i]);
    }

    /* Free the memory allocated to pointers gpu_arr1, gpu_arr2, gpu_arr3 */
    hipFree(gpu_arr1);
    hipFree(gpu_arr2);
    hipFree(gpu_arr3);

    return 0;
}
